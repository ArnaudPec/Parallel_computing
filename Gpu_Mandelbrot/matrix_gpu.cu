#include "hip/hip_runtime.h"
#include <stdio.h>
#define N 22*40

__global__ void addMatrix(float *a, float *b, float *c){

    long idx = blockIdx.x*blockDim.x+threadIdx.x;
    long idy = blockIdx.y*blockDim.y+threadIdx.y;
    //long offset = idx+idy * N;
    long offset = idy+idx * N;
    c[offset]=a[offset]+b[offset];
}

int main(){
	

	float a[N*N];
	float b[N*N];
	float c[N*N];
    long size = N*N*sizeof(float);
    float *cd;
    float *ad;
    float *bd;
    float theTime;

    hipEvent_t myEvent, laterEvent;
    hipEventCreate(&myEvent);
    hipEventCreate(&laterEvent);

    hipMalloc((void **)&cd, size); 
    hipMalloc((void **)&ad, size); 
    hipMalloc((void **)&bd, size); 

	for (long i = 0; i < N; i++)
		for (long j = 0; j < N; j++)
		{
			a[i+j*N] = 10 + i;
			b[i+j*N] = (float)j / N;
		}
    hipMemcpy(ad,a,size,hipMemcpyHostToDevice);
    hipMemcpy(bd,b,size,hipMemcpyHostToDevice);
    hipMemcpy(cd,c,size,hipMemcpyHostToDevice);

	dim3 dimBlock( 22, 22);
	dim3 dimGrid( N/22, N/22 );
/*
	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			printf("%0.2f ", a[i+j*N]);
		}
		printf("\n");
	}

    printf("\n");

	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			printf("%0.2f ", b[i+j*N]);
		}
		printf("\n");
	}

    printf("\ngpu calc\n");
*/
    hipEventRecord(myEvent, 0);
	addMatrix <<< dimGrid, dimBlock >>>(ad,bd,cd);
    hipDeviceSynchronize();

    hipEventRecord(laterEvent, 0);
    
	hipMemcpy( c, cd, size, hipMemcpyDeviceToHost ); 

    hipEventSynchronize(laterEvent); 
    hipEventElapsedTime(&theTime, myEvent, laterEvent);

	for (long i = 0; i < N; i++)
	{
		for (long j = 0; j < N; j++)
		{
			printf("%0.2f ", c[i+j*N]);
		}
		printf("\n");
	}

	printf("%f ms\n", theTime);

    hipEventDestroy(laterEvent);
    hipEventDestroy(myEvent);

    hipFree(cd);
    hipFree(ad);
    hipFree(bd);

    return 0;
}

/*
 @see addMatrix

 2. It compiles, but it doesn't run because of hw limitations(nowadays std is
 1024 threads per block)
   */
