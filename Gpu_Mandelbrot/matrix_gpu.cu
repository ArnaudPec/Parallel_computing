#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void addMatrix(float *a, float *b, float *c){

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int offset = idx+idy;
    c[offset]=a[offset]+b[offset];
}

int main(){
	
	const int N = 16;

	float a[N*N];
	float b[N*N];
	float c[N*N];
    int size = N*N*sizeof(float);
    float *cd;
    float *ad;
    float *bd;
    float theTime;

    hipEvent_t myEvent, laterEvent;
    hipEventCreate(&myEvent);
    hipEventCreate(&laterEvent);
    hipEventRecord(myEvent, 0);

    hipMalloc((void **)&cd, size); 
    hipMalloc((void **)&ad, size); 
    hipMalloc((void **)&bd, size); 

    hipMemcpy(ad,a,size,hipMemcpyHostToDevice);
    hipMemcpy(bd,b,size,hipMemcpyHostToDevice);
    hipMemcpy(cd,c,size,hipMemcpyHostToDevice);

	dim3 dimBlock( N, N );
	dim3 dimGrid( 1, 1 );

	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			a[i+j*N] = 10 + i;
			b[i+j*N] = (float)j / N;
		}

	addMatrix <<< dimGrid, dimBlock >>>(ad,bd,cd);
    hipDeviceSynchronize();

	hipMemcpy( c, cd, size, hipMemcpyDeviceToHost ); 

    hipEventRecord(laterEvent, 0);
    hipEventSynchronize(laterEvent); 
    hipEventElapsedTime(&theTime, myEvent, laterEvent);

	for (int i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			printf("%0.2f ", c[i+j*N]);
		}
		printf("\n");
	}

	printf("%3.1f ms\n", theTime);

    hipEventDestroy(&laterEvent);
    hipEventDestroy(&myEvent);

    hipFree(cd);
    hipFree(ad);
    hipFree(bd);

    return 0;
}

/*
 @see addMatrix

 2. It compiles, but it doesn't run because of hw limitations(nowadays std is
 1024 threads per block)
   */
