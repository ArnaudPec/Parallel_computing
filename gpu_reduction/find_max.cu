#include "hip/hip_runtime.h"
// Reduction lab, find maximum

#include <stdio.h>
#include "milli.c"

const int tx = 8;
const int ty = 1;
const int threads_per_block = tx * ty;
const int gx = 8;
const int gy = 1;
const int blocks_per_grid = gx * gy;
__global__ void find_max(int *data, int N)
{
	__shared__ int cache[threads_per_block];
	int tid = threadIdx.x;
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	cache[tid] = data[i];
	__syncthreads();

	for (int s = blockDim.x/2; s > 0; s>>=1)
	{
		if(tid < s)
		{
			cache[tid] ++;
			//if(cache[tid] < cache[tid + s])
			//	cache[tid] = cache[tid + s];
		}
		__syncthreads();
	}
	data[i] = cache[tid];

}

void launch_cuda_kernel(int *data, int N)
{
	// Handle your CUDA kernel launches in this function
	
	int *devdata;
	int size = sizeof(int) * N;
	hipMalloc( (void**)&devdata, size);
	hipMemcpy(devdata, data, size, hipMemcpyHostToDevice );
	
	// Dummy launch
	dim3 dimBlock( tx, ty );
	dim3 dimGrid( gx, gy );
	find_max<<<dimGrid, dimBlock>>>(devdata, N);
	hipError_t err = hipPeekAtLastError();
	if (err) printf("hipPeekAtLastError %d %s\n", err, hipGetErrorString(err));

	// Only the result needs copying!
	hipMemcpy(data, devdata, size, hipMemcpyDeviceToHost ); 
	int i;
	for(i = 0; i < N; i++)
	{
		printf("%i\n", data[i]);
	}
	hipFree(devdata);
}

// CPU max finder (sequential)
void find_max_cpu(int *data, int N)
{
  int i, m;
  
	m = data[0];
	for (i=0;i<N;i++) // Loop over data
	{
		if (data[i] > m)
			m = data[i];
	}
	data[0] = m;
}

//#define SIZE 1024
#define SIZE 16
// Dummy data in comments below for testing
int data[] = {1, 2, 5, 3, 6, 8, 5, 3, 1, 65, 8, 5, 3, 34, 2, 54};
int data2[] = {1, 2, 5, 3, 6, 8, 5, 3, 1, 65, 8, 5, 3, 34, 2, 54};

int main()
{
  // Generate 2 copies of random data
  /*
  srand(time(NULL));
  for (long i=0;i<SIZE;i++)
  {
    data[i] = rand() % (SIZE * 5);
    data2[i] = data[i];
  }
  */
  
  // The GPU will not easily beat the CPU here!
  // Reduction needs optimizing or it will be slow.
  ResetMilli();
  find_max_cpu(data, SIZE);
  printf("CPU time %f\n", GetSeconds());
  ResetMilli();
  launch_cuda_kernel(data2, SIZE);
  printf("GPU time %f\n", GetSeconds());

  // Print result
  printf("\n");
  printf("CPU found max %d\n", data[0]);
  printf("GPU found max %d\n", data2[0]);
}
